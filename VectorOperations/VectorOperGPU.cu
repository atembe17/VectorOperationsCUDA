#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <stdio.h>
#include "common.cuh"
#include "math.h"

//Const to set TILE_SIZE of the device
const float TILE_SIZE = 1024;

//Kernel method for vector addition
__global__ void VectoraddKernel(float* Agpu, float* Bgpu, float* Cgpu, int size)
{
	//Thread id
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < size) {
		Cgpu[tid] = Agpu[tid] + Bgpu[tid];
	}
}

//Kernel method for vector subtraction
__global__ void VectorsubtractKernel(float* Agpu, float* Bgpu, float* Cgpu, int size)
{
	//Thread id
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < size) {
		Cgpu[tid] = Agpu[tid] - Bgpu[tid];
	}
}

//Kernel method for vector scaling
__global__ void VectorscaleKernel(float* Agpu, float* Cgpu, float scaling, int size)
{
	//Thread id
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < size) {
		Cgpu[tid] = Agpu[tid] * scaling;
	}
}

// Method to allocate memory and invoke kernel method for vector addition
bool addVectorGPU(float* M, float* N, float* P, int size) {
	int bytes = size * sizeof(float);
	float* Agpu, * Bgpu, * Cgpu;
	//Page lock memory mapping
	hipHostGetDevicePointer((void**)&Agpu, M, 0);
	hipHostGetDevicePointer((void**)&Bgpu, N, 0);
	hipHostGetDevicePointer((void**)&Cgpu, P, 0);
	//Set the block and grid dimens
	dim3 dimBlock(TILE_SIZE);
	dim3 dimGrid((int)ceil((float)size / (float)TILE_SIZE));
	// Launch the kernel on a size-by-size block of threads
	VectoraddKernel << <dimGrid, dimBlock >> > (Agpu, Bgpu, Cgpu, size);
	hipDeviceSynchronize();
	//Return error if any 
	hipError_t status = hipGetLastError();
	if (status != hipSuccess) {
		printf("Kernel failed: %s", hipGetErrorString(status));
		return false;
	}
	return true;
}

// Method to allocate memory and invoke kernel method for vector subtraction
bool subtractVectorGPU(float* M, float* N, float* P, int size) {
	int bytes = size * sizeof(float);
	float* Agpu, * Bgpu, * Cgpu;
	//Page lock memory mapping
	hipHostGetDevicePointer((void**)&Agpu, M, 0);
	hipHostGetDevicePointer((void**)&Bgpu, N, 0);
	hipHostGetDevicePointer((void**)&Cgpu, P, 0);
	//Set the block and grid dimens
	dim3 dimBlock(TILE_SIZE);
	dim3 dimGrid((int)ceil((float)size / (float)TILE_SIZE));
	// Launch the kernel on a size-by-size block of threads
	VectorsubtractKernel << <dimGrid, dimBlock >> > (Agpu, Bgpu, Cgpu, size);
	hipDeviceSynchronize();
	//Return error if any 
	hipError_t status = hipGetLastError();
	if (status != hipSuccess) {
		printf("Kernel failed: %s", hipGetErrorString(status));
		return false;
	}

	return true;
}

// Method to allocate memory and invoke kernel method for vector scaling
bool scaleVectorGPU(float* M, float* P, float scaling, int size) {
	int bytes = size * sizeof(float);
	float* Agpu, * Cgpu;
	//Page lock memory mapping
	hipHostGetDevicePointer((void**)&Agpu, M, 0);
	hipHostGetDevicePointer((void**)&Cgpu, P, 0);
	//Set the block and grid dimens
	dim3 dimBlock(TILE_SIZE);
	dim3 dimGrid((int)ceil((float)size / (float)TILE_SIZE));
	// Launch the kernel on a size-by-size block of threads
	VectorscaleKernel << <dimGrid, dimBlock >> > (Agpu, Cgpu, scaling, size);
	hipDeviceSynchronize();
	//Return error if any 
	hipError_t status = hipGetLastError();
	if (status != hipSuccess) {
		printf("Kernel failed: %s", hipGetErrorString(status));
		return false;
	}

	return true;
}
